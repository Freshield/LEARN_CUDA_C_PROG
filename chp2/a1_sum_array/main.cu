
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <stdio.h>
#include <hip/driver_types.h>

void sumArrayOnHost(float *A, float *B, float *C, const int N){
    for (int idx=0; idx < N; idx++){
        C[idx] = A[idx] + B[idx];
    }
}

void initialData(float *ip, int size){
    time_t t;
    srand((unsigned int) time(&t));

    for (int i=0; i<size; i++){
        ip[i] = (float)(rand() & 0xFF)/10.0f;
    }
}

int main(int argc, char **argv){
    int nElem = 1024;
    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B, *h_C;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    h_C = (float *)malloc(nBytes);

    initialData(h_A, nElem);
    initialData(h_B, nElem);

    sumArrayOnHost(h_A, h_B, h_C, nElem);

    for (int i=0; i<nElem; i++){
        printf("%f\n",h_C[i]);
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((float **)&d_A, nBytes);
    hipMalloc((float **)&d_B, nBytes);
    hipMalloc((float **)&d_C, nBytes);

    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    hipFree(d_A);
    hipFree(d_B);

    free(h_A);
    free(h_B);
    free(h_C);

    return(0);
}